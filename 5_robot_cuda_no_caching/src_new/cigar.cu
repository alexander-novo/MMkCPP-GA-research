#include <errno.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/stat.h>
#include <unistd.h>

#include "case.cuh"
#include "hip/hip_runtime.h"
#include ""
#include "init.cuh"
#include "type.cuh"

// extern int errno;

void Statistics(IPTR, Population *p);
void Report(int gen, IPTR pop, Population *p);

void Initialize(int argc, char *argv[], Population *p, Functions *f);

void WritePid(char *pidFile);
void RmPidFile(char *pidFile);

void PhenoPrint(FILE *fp, IPTR pop, Population *p);  // modified

int main(int argc, char *argv[]) {
	IPTR tmp;            // used for swapping two IPTRs
	int foo;             /* just a placeholder for a value that is not used */
	Population pop, *p;  // The current population under inspection
	Functions funcs,
	    *f;  // A set of function pointers which are swapped out depending on the parameters.

	p = &pop;
	f = &funcs;

	p->generation = 0;

	printf("Blah\n");
	fflush(stdout);
	Initialize(argc, argv, p, f);

	// WritePid(p->pidFile);

	printf("Blah\n");
	fflush(stdout);
	while (p->generation < p->maxgen) {
		p->generation++;

		foo = f->CurrentGA(p->oldpop, p->newpop, p->generation, p, f);

		if (p->injectFraction > 0.0) {
			if ((p->generation % p->injectPeriod == 0) && (p->generation <= p->injectStop)) {
				LoadCases(p->newpop, p->generation, p->injectFraction, p, f);
				/* printf("Loaded cases %d\n", (int) (loadPerc/100.0 * popsize));*/
			}
		}
		Statistics(p->newpop, p);
		Report(p->generation, p->newpop, p);

		// Record data (best individual at each gen)
		FILE *dataFile;

		dataFile = fopen("myData.txt", "a");

		fprintf(dataFile, "%f\n", p->newpop[p->maxi].objfunc);

		fclose(dataFile);

		// Record best route
		FILE *routeFile;
		routeFile = fopen("myRoutes.txt", "a");

		// PhenoPrint<<<1,1>>>(routeFile, p->newpop, p);
		PhenoPrint(routeFile, p->newpop, p);
		// Wait for GPU to finish before accessing on host
		// hipDeviceSynchronize();

		fprintf(routeFile, "\n");
		for (int i = 0; i < p->newpop->chromLen; i++)
			fprintf(routeFile, "%d, ", p->newpop->chrom[i]);
		fprintf(routeFile, "\n");
		fclose(routeFile);

		tmp       = p->oldpop;
		p->oldpop = p->newpop;
		p->newpop = tmp;
	}
	if (p->nCurrentCases > 0) {
		p->nCases = FindNCases(p->nCFile);
		StoreNcases(p->nCFile, p->nCases, p->nCurrentCases);
	}
	// RmPidFile(p->pidFile);

	return 0;
}

void WritePid(char *fname) {
	struct stat buf;
	int er;
	FILE *fp;

	er = stat(fname, &buf);
	if (!(er == -1 || errno == ENOENT)) {
		fprintf(stderr, "Lock file (%s) exists, Process running\n", fname);
		fprintf(stderr, "This process is exiting....\n");
		exit(1);
	}

	if ((fp = fopen(fname, "w")) == NULL) {
		fprintf(stderr, "Error in opening file %s for writing\n", fname);
		exit(2);
	}

	// fprintf(fp, "%lu", getpid());
}

void RmPidFile(char *fname) {
	unlink(fname);
}

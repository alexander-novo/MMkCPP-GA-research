#include "hip/hip_runtime.h"
﻿#include <ctype.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>

#include <algorithm>

#include "hip/hip_runtime.h"
#include ""
#include "init.cuh"
#include "random.cuh"
#include "type.cuh"
#include "utils.cuh"

#define INF 0x7f7f7f7f

Coords* cities;

// initialize global tables in host function
int graph_h[NUMBER_POINTS][NUMBER_POINTS];
Edges edges_h[NUMBER_EDGES];
int dijkstraDistTable_h[NUMBER_POINTS][NUMBER_POINTS];

__constant__ int graph[NUMBER_POINTS * NUMBER_POINTS];
__constant__ Edges edges[NUMBER_EDGES];
int edgeValues[NUMBER_EDGES];

__device__ int dijkstraRouteTable[NUMBER_POINTS * NUMBER_POINTS * MAX_ROUTE_LENGTH];
////[NUMBER_POINTS][MAX_ROUTE_LENGTH];   //with very large graphs this may exceed memory limits

__device__ int dijkstraDistTable[NUMBER_POINTS * NUMBER_POINTS];

// bool DIJKSTRA_INITIALIZED_FLAG = false;

// To handle deadheading vs. inspection traversal
// double visitedGraph[NUMBER_POINTS][NUMBER_POINTS];
// double verifyGraph[NUMBER_POINTS][NUMBER_POINTS];

__constant__ const double DEADHEADING_RATIO =
    1.0;  // Ratio of deadheading cost to service cost (can be adjusted to reflect real speeds of
          // robot)
__constant__ const int NUM_PERMUTATIONS = 1;

// int DIJKSTRA_COUNT = 0; //count then number of times Dikstra's algorithm was called
int INIT_COUNT = 0;

// extern "C" double Eval(IPTR pj);
// extern "C" double TSPDist(ChromType c1, ChromType c2);

int totalLength;

typedef struct {  // struct to return cost and route of vertices at once in Dijkstra
	int dist;
	int route[MAX_ROUTE_LENGTH];

} Tuple;

__device__ int Dijkstra(int start, int end, int route[MAX_ROUTE_LENGTH],
                        int visitedGraph[NUMBER_POINTS][NUMBER_POINTS]);
double TSPDist(ChromType c1, ChromType c2);
void RemoveCrossings(IPTR pj, int start, int end);
void Reverse(IPTR tmp, int lchrom, ChromType c1, ChromType c2);
__device__ Tuple routeDistance(IPTR pj, int posStart, int posEnd, int* currentPoint,
                               int visitedGraph[NUMBER_POINTS][NUMBER_POINTS], int* chroms,
                               int c_index);
__device__ Tuple PhenoRouteGet(IPTR pj, int posStart, int posEnd, int* currentPoint,
                               int visitedGraph[NUMBER_POINTS][NUMBER_POINTS],
                               int verifyGraph[NUMBER_POINTS][NUMBER_POINTS], int* chroms,
                               int c_index);

__device__ int Dijkstra(int start, int end, int route[MAX_ROUTE_LENGTH],
                        int visitedGraph[NUMBER_POINTS][NUMBER_POINTS]) {
	// printf("entered dijkstra\n");
	// Check cache table of dijkstra route values first
	// commented out to temporarily disable caching
	if (dijkstraDistTable[start * NUMBER_POINTS + end] > -1) {
		for (int i = 0; i < MAX_ROUTE_LENGTH; i++) {
			route[i] = dijkstraRouteTable[(start * NUMBER_POINTS + end) * MAX_ROUTE_LENGTH + i];
		}
		return dijkstraDistTable[start * NUMBER_POINTS + end];
	} else {
		// printf("entered dijkstra else\n");
		// printf("Dijkstra long\n");
		// DIJKSTRA_COUNT += 1;
		// if value not in table, compute it for the first time
		int dis[NUMBER_POINTS];
		bool vis[NUMBER_POINTS];

		int prev[NUMBER_POINTS];  // to track route

		memset(dis, INF, sizeof(int) * NUMBER_POINTS);
		memset(vis, false, sizeof(bool) * NUMBER_POINTS);
		memset(prev, -1, sizeof(int) * NUMBER_POINTS);

		dis[start] = 0;
		vis[start] = true;

		// update the connected distance
		for (int j = 0; j < NUMBER_POINTS; j++) {
			if (vis[j] == false && graph[start * NUMBER_POINTS + j] > 0) {
				dis[j] = graph[start * NUMBER_POINTS + j] *
				         visitedGraph[start][j];  // Multiply distance by 1.0 if unvisited, by
				                                  // deadheading ratio if visited --
			}
		}

		for (int i = 0; i < NUMBER_POINTS; i++) {
			// find the connected point with the shortest distance
			int minx    = INF;
			int minmark = 0;
			for (int j = 0; j < NUMBER_POINTS; j++) {
				if (vis[j] == false && dis[j] <= minx) {
					minx    = dis[j];
					minmark = j;
				}
			}
			// mark the point
			vis[minmark] = true;

			// update all the unmarked points connected to the current marked point.
			for (int j = 0; j < NUMBER_POINTS; j++) {
				if (vis[j] == false && graph[minmark * NUMBER_POINTS + j] > 0 &&
				    dis[j] >
				        dis[minmark] + (graph[minmark * NUMBER_POINTS + j] *
				                        visitedGraph[minmark][j]))  // mult with visited graph to
				                                                    // account for deadheading --
				{
					dis[j] =
					    dis[minmark] + (graph[minmark * NUMBER_POINTS + j] *
					                    visitedGraph[minmark][j]);  // mult with visited graph to
					                                                // account for deadheading --
					prev[j] = minmark;
				}
			}
		}

		// prepare route array
		for (int i = 0; i < MAX_ROUTE_LENGTH; i++) { route[i] = -1; }

		// Reconstruct route (produces route in reverse order: destination to source)
		int u          = end;
		int routeIndex = 0;
		if (prev[u] != -1 || u == start) {
			while (u != -1) {
				route[routeIndex] = u;
				routeIndex++;
				u = prev[u];
			}
		}
		for (int i = 0; i < MAX_ROUTE_LENGTH; i++) {
			if (route[i] < 0) { route[i] = -1; }
		}

		// copy finalized route into cache table
		// temporarily commented out to disable caching
		for (int i = 0; i < MAX_ROUTE_LENGTH; i++) {
			// printf("%d\n", route[i]);
			dijkstraRouteTable[(start * NUMBER_POINTS + end) * MAX_ROUTE_LENGTH + i] = route[i];
		}

		// copy finalized distance into cache table
		dijkstraDistTable[start * NUMBER_POINTS + end] = dis[end];

		return dis[end];
	}
}

__host__ int Dijkstra_host(int start, int end, int route[MAX_ROUTE_LENGTH],
                           int visitedGraph[NUMBER_POINTS][NUMBER_POINTS]) {
	// printf("\n d_host entered");

	// Check cache table of dijkstra route values first
	// commented out to temporarily disable caching

	// if (dijkstraDistTable[start * NUMBER_POINTS + end] > -1) {
	// 	for (int i = 0; i < MAX_ROUTE_LENGTH; i++) {
	// 		route[i] = dijkstraRouteTable[(start * NUMBER_POINTS + end) * MAX_ROUTE_LENGTH + i];
	// 	}
	// 	return dijkstraDistTable[start * NUMBER_POINTS + end];

	// } else {
	// printf("Dijkstra long\n");
	// DIJKSTRA_COUNT += 1;
	// if value not in table, compute it for the first time
	int dis[NUMBER_POINTS];
	bool vis[NUMBER_POINTS];

	int prev[NUMBER_POINTS];  // to track route

	memset(dis, INF, sizeof(int) * NUMBER_POINTS);
	memset(vis, false, sizeof(bool) * NUMBER_POINTS);
	memset(prev, -1, sizeof(int) * NUMBER_POINTS);

	dis[start] = 0;
	vis[start] = true;

	// printf("\n d_host 1");

	// update the connected distance
	for (int j = 0; j < NUMBER_POINTS; j++) {
		if (vis[j] == false && graph_h[start][j] > 0) {
			dis[j] = graph_h[start][j] *
			         visitedGraph[start][j];  // Multiply distance by 1.0 if unvisited, by
			                                  // deadheading ratio if visited --
		}
	}

	// printf("\n d_host 2");

	for (int i = 0; i < NUMBER_POINTS; i++) {
		// find the connected point with the shortest distance
		int minx    = INF;
		int minmark = 0;
		for (int j = 0; j < NUMBER_POINTS; j++) {
			if (vis[j] == false && dis[j] <= minx) {
				minx    = dis[j];
				minmark = j;
			}
		}
		// mark the point
		vis[minmark] = true;

		// update all the unmarked points connected to the current marked point.
		for (int j = 0; j < NUMBER_POINTS; j++) {
			if (vis[j] == false && graph_h[minmark][j] > 0 &&
			    dis[j] > dis[minmark] + (graph_h[minmark][j] *
			                             visitedGraph[minmark][j]))  // mult with visited graph to
			                                                         // account for deadheading --
			{
				dis[j] = dis[minmark] + (graph_h[minmark][j] *
				                         visitedGraph[minmark][j]);  // mult with visited graph to
				                                                     // account for deadheading --
				prev[j] = minmark;
			}
		}
	}

	// printf("\n d_host 3");

	// prepare route array
	for (int i = 0; i < MAX_ROUTE_LENGTH; i++) { route[i] = -1; }

	// printf("\n d_host 4");
	// Reconstruct route (produces route in reverse order: destination to source)
	int u          = end;
	int routeIndex = 0;
	if (prev[u] != -1 || u == start) {
		while (u != -1) {
			route[routeIndex] = u;
			routeIndex++;
			u = prev[u];
		}
	}

	// printf("\n d_host 5");
	for (int i = 0; i < MAX_ROUTE_LENGTH; i++) {
		if (route[i] < 0) { route[i] = -1; }
	}

	// copy finalized route into cache table

	// Don't write anything as hsot
	// for (int i = 0; i < MAX_ROUTE_LENGTH; i++) {
	// 	// printf("%d\n", route[i]);
	// 	dijkstraRouteTable[(start * NUMBER_POINTS + end) * MAX_ROUTE_LENGTH + i] = route[i];
	// }

	// copy finalized distance into cache table

	// Don't write anything as the host
	// dijkstraDistTable[start * NUMBER_POINTS + end] = dis[end];
	// printf("\nd_host returns: %d", dis[end]);
	return dis[end];
	//}
}

// GPU device-friendly max function for integers (can't call a const expr host function from a
// device function)
__device__ __host__ int g_max(int a, int b) {
	if (a >= b) {
		return a;
	} else {
		return b;
	}
}

__device__ int getStartPoint(IPTR pj, int p0, int p1,
                             int visitedGraph[NUMBER_POINTS][NUMBER_POINTS], int* chroms,
                             int c_index) {  // p0=30, p1=31   -- returning too high value
	// printf("getStartPoint\n");
	int p0s = edges[chroms[c_index * pj->chromLen + p0]]
	              .start;  // p0s = start point for edge. p0e = end point for edge
	int p0e = edges[chroms[c_index * pj->chromLen + p0]].end;
	int p1s = edges[chroms[c_index * pj->chromLen + p1]].start;
	int p1e = edges[chroms[c_index * pj->chromLen + p1]]
	              .end;  // posibly going past the max length since robots index >= 83 when
	                     // asked for robot+1 and robot+2

	if (p0s == p1s || p0s == p1e)  // if 0s=1s or 0s=1e
	{
		return p0e;
	} else if (p0e == p1s || p0e == p1e)  // if 0e=1s or 0e=1e
	{
		return p0s;
	} else {
		int route[MAX_ROUTE_LENGTH];
		int d00 = Dijkstra(p0s, p1s, route, visitedGraph);  // 0start to 1start
		int d01 = Dijkstra(p0s, p1e, route, visitedGraph);  // 0start to 1end
		int d10 = Dijkstra(p0e, p1s, route, visitedGraph);  // 0e to 1s
		int d11 = Dijkstra(p0e, p1e, route, visitedGraph);  // 0e to 1e
		int max = g_max(g_max(g_max(d00, d01), d10), d11);  // shouldn't this be min?
		if (d00 == max || d01 == max) {  // if 0s - 1s or 1e is longest start at 0s
			return p0s;
		} else if (d10 == max || d11 == max) {  // if 0e - 1s/1e is longest start at 0e
			return p0e;
		}
	}
	return p0e;
	// printf("getStartPoint ending\n");
}

__host__ int getStartPoint_host(
    IPTR pj, int p0, int p1,
    int visitedGraph[NUMBER_POINTS][NUMBER_POINTS]) {  // p0=30, p1=31   -- returning too high value
	// printf("getStartPoint\n");
	int p0s = edges_h[pj->chrom[p0]].start;  // p0s = start point for edge. p0e = end point for edge
	int p0e = edges_h[pj->chrom[p0]].end;
	int p1s = edges_h[pj->chrom[p1]].start;
	int p1e = edges_h[pj->chrom[p1]].end;  // posibly going past the max length since robots index
	                                       // >= 83 when asked for robot+1 and robot+2

	if (p0s == p1s || p0s == p1e)  // if 0s=1s or 0s=1e
	{
		return p0e;
	} else if (p0e == p1s || p0e == p1e)  // if 0e=1s or 0e=1e
	{
		return p0s;
	} else {
		int route[MAX_ROUTE_LENGTH];
		int d00 = Dijkstra_host(p0s, p1s, route, visitedGraph);  // 0start to 1start
		int d01 = Dijkstra_host(p0s, p1e, route, visitedGraph);  // 0start to 1end
		int d10 = Dijkstra_host(p0e, p1s, route, visitedGraph);  // 0e to 1s
		int d11 = Dijkstra_host(p0e, p1e, route, visitedGraph);  // 0e to 1e
		int max = g_max(g_max(g_max(d00, d01), d10), d11);       // shouldn't this be min?
		if (d00 == max || d01 == max) {  // if 0s - 1s or 1e is longest start at 0s
			return p0s;
		} else if (d10 == max || d11 == max) {  // if 0e - 1s/1e is longest start at 0e
			return p0e;
		}
	}
	return p0e;
	// printf("getStartPoint ending\n");
}

// Eval one depot starting at point 0
/*
double Eval1R(IPTR pj)
{
    int currentPoint1 = getStartPoint(pj, 0, 1);
    //route distance from 0 to the end
    return routeDistance(pj, 0, pj->chromLen, &currentPoint1).dist;
}*/

__device__ Tuple routeDistance(IPTR pj, int posStart, int posEnd, int* currentPoint,
                               int visitedGraph[NUMBER_POINTS][NUMBER_POINTS], int* chroms,
                               int c_index) {
	// printf("routeDistance\n");

	// store vertices of route in this array
	// int* myRoute;
	int myRoute[MAX_ROUTE_LENGTH];
	int dummyRoute[MAX_ROUTE_LENGTH];
	int dummyRoute2[MAX_ROUTE_LENGTH];

	for (int z = 0; z < MAX_ROUTE_LENGTH; z++) {
		myRoute[z]     = -1;
		dummyRoute[z]  = -1;
		dummyRoute2[z] = -1;
	}
	// myRoute = myChars;
	int routeIndex = 0;

	int currentDistance = 0;
	// first edge
	if (*currentPoint == edges[chroms[c_index * pj->chromLen + posStart]].start) {
		*currentPoint = edges[chroms[c_index * pj->chromLen + posStart]].end;
		// Commented out because it doesn't do anything?
		// graph[edges[chroms[c_index * pj->chromLen + posStart]].start * NUMBER_POINTS +
		//       edges[chroms[c_index * pj->chromLen + posStart]].end];

		/*
		myRoute[routeIndex] = edges[pj->chrom[posStart]].start;
		routeIndex++;
		myRoute[routeIndex] = edges[pj->chrom[posStart]].end;
		routeIndex++;*/
	} else if (*currentPoint == edges[chroms[c_index * pj->chromLen + posStart]].end) {
		*currentPoint = edges[chroms[c_index * pj->chromLen + posStart]].start;
		// No effectr
		// graph[edges[chroms[c_index * pj->chromLen + posStart]].start * NUMBER_POINTS +
		//       edges[chroms[c_index * pj->chromLen + posStart]].end];

		/*
		myRoute[routeIndex] = edges[pj->chrom[posStart]].end;
		routeIndex++;
		myRoute[routeIndex] = edges[pj->chrom[posStart]].start;
		routeIndex++;*/
	}

	// calcluate the distance of robot 1 from 2nd edge to robot 2
	for (int i = posStart + 1; i < posEnd; i++) {
		int dissta = Dijkstra(*currentPoint, edges[chroms[c_index * pj->chromLen + i]].start,
		                      dummyRoute, visitedGraph);
		int disend = Dijkstra(*currentPoint, edges[chroms[c_index * pj->chromLen + i]].end,
		                      dummyRoute2, visitedGraph);
		if (dissta < disend) {
			currentDistance += dissta;
			*currentPoint = edges[chroms[c_index * pj->chromLen + i]].end;

			for (int z = 0; z < MAX_ROUTE_LENGTH; z++) {
				if (dummyRoute[z] != -1) {
					myRoute[routeIndex] = dummyRoute[z];
					routeIndex++;
				}
			}

			for (int z = 0; z < MAX_ROUTE_LENGTH - 1; z++) {
				if (dummyRoute[z] != -1 && dummyRoute[z + 1] != -1) {
					// visitedGraph[dummyRoute[z]][dummyRoute[z + 1]] = DEADHEADING_RATIO;
					// //mark edge as visited visitedGraph[dummyRoute[z + 1]][dummyRoute[z]] =
					// DEADHEADING_RATIO;
				}
			}

		} else {
			currentDistance += disend;
			*currentPoint = edges[chroms[c_index * pj->chromLen + i]].start;

			for (int z = 0; z < MAX_ROUTE_LENGTH; z++) {
				if (dummyRoute2[z] != -1) {
					myRoute[routeIndex] = dummyRoute2[z];
					routeIndex++;
				}
			}

			for (int z = 0; z < MAX_ROUTE_LENGTH - 1; z++) {
				if (dummyRoute2[z] != -1 && dummyRoute2[z + 1] != -1) {
					// visitedGraph[dummyRoute2[z]][dummyRoute2[z + 1]] = DEADHEADING_RATIO;
					// //mark edge as visited visitedGraph[dummyRoute2[z + 1]][dummyRoute2[z]] =
					// DEADHEADING_RATIO;
				}
			}
		}
		currentDistance +=
		    graph[edges[chroms[c_index * pj->chromLen + i]].start * NUMBER_POINTS +
		          edges[chroms[c_index * pj->chromLen + i]].end] *
		    visitedGraph[edges[chroms[c_index * pj->chromLen + i]].start]
		                [edges[chroms[c_index * pj->chromLen + i]]
		                     .end];  // mult with visited graph to account for deadheading --
		// visitedGraph[edges[pj->chrom[i]].start][edges[pj->chrom[i]].end] = DEADHEADING_RATIO;
		// //mark edge as visited
		// visitedGraph[edges[pj->chrom[i]].end][edges[pj->chrom[i]].start] = DEADHEADING_RATIO;
		// //mark edge as visited in reverse direction

		for (int z = 0; z < MAX_ROUTE_LENGTH; z++) {
			dummyRoute[z]  = -1;
			dummyRoute2[z] = -1;
		}

		/*
		if (i == posStart + 1)
		{
		    myRoute[routeIndex] = edges[pj->chrom[i]].start;
		    routeIndex++;
		}*/

		myRoute[routeIndex] = edges[chroms[c_index * pj->chromLen + i]].start;
		routeIndex++;

		myRoute[routeIndex] = edges[chroms[c_index * pj->chromLen + i]].end;
		routeIndex++;
	}
	// return currentDistance;

	Tuple result;
	result.dist = currentDistance;

	for (int z = 0; z < MAX_ROUTE_LENGTH; z++) { result.route[z] = myRoute[z]; }

	// clean route of duplicates
	for (int x = 0; x < 2; x++) {
		for (int i = 1; i < MAX_ROUTE_LENGTH; i++) {
			if (result.route[i - 1] == result.route[i]) {
				for (int j = i; j < MAX_ROUTE_LENGTH - 1; j++) {
					result.route[j] = result.route[j + 1];
				}
			}
		}
	}

	/*
	printf("\nroute: ");
	for (int z = 0; z < MAX_ROUTE_LENGTH; z++)
	{
	    if (result.route[z] != -1)
	    {
	        printf("%d, ", result.route[z]);
	    }
	}
	printf("\n");*/

	// printf("routeDistance ending\n");

	return result;
}

__host__ void getRobot12_host(IPTR pj, int* robot1, int* robot2, int* robot3, int* robot4,
                              int* robot5) {  // modified
	for (int i = 0; i < pj->chromLen; i++) {
		if (pj->chrom[i] == pj->chromLen - 5) { *robot1 = i; }
		if (pj->chrom[i] == pj->chromLen - 4) { *robot2 = i; }
		if (pj->chrom[i] == pj->chromLen - 3) { *robot3 = i; }
		if (pj->chrom[i] == pj->chromLen - 2) {
			*robot4 = i;
		} else if (pj->chrom[i] == pj->chromLen - 1) {
			*robot5 = i;
		}
	}
	// make sure the order is 0----robot1-----robot2-3-4-5--82
	int tmp;
	for (int i = 0; i < 4; i++) {  // sort values to correct positons
		if (*robot1 > *robot2) {
			tmp     = *robot1;
			*robot1 = *robot2;
			*robot2 = tmp;
		}
		if (*robot2 > *robot3) {
			tmp     = *robot2;
			*robot2 = *robot3;
			*robot3 = tmp;
		}
		if (*robot3 > *robot4) {
			tmp     = *robot3;
			*robot3 = *robot4;
			*robot4 = tmp;
		}
		if (*robot4 > *robot5) {
			tmp     = *robot4;
			*robot4 = *robot5;
			*robot5 = tmp;
		}
	}
}

__device__ void getRobot12_device(IPTR pj, int* robot1, int* robot2, int* robot3, int* robot4,
                                  int* robot5, int* chroms, int c_index) {  // modified
	for (int i = 0; i < pj->chromLen; i++) {
		if (chroms[pj->chromLen * c_index + i] == pj->chromLen - 5) { *robot1 = i; }
		if (chroms[pj->chromLen * c_index + i] == pj->chromLen - 4) { *robot2 = i; }
		if (chroms[pj->chromLen * c_index + i] == pj->chromLen - 3) { *robot3 = i; }
		if (chroms[pj->chromLen * c_index + i] == pj->chromLen - 2) {
			*robot4 = i;
		} else if (chroms[pj->chromLen * c_index + i] == pj->chromLen - 1) {
			*robot5 = i;
		}
	}
	// make sure the order is 0----robot1-----robot2-3-4-5--82
	int tmp;
	for (int i = 0; i < 4; i++) {  // sort values to correct positons
		if (*robot1 > *robot2) {
			tmp     = *robot1;
			*robot1 = *robot2;
			*robot2 = tmp;
		}
		if (*robot2 > *robot3) {
			tmp     = *robot2;
			*robot2 = *robot3;
			*robot3 = tmp;
		}
		if (*robot3 > *robot4) {
			tmp     = *robot3;
			*robot3 = *robot4;
			*robot4 = tmp;
		}
		if (*robot4 > *robot5) {
			tmp     = *robot4;
			*robot4 = *robot5;
			*robot5 = tmp;
		}
	}
}

// Eval 5 robots starting 0-82 edges, 83, 84, 85, 86, 87 are 5 robots
__device__ double Eval(IPTR pj, int* chroms, int c_index)  // modified 83-87 are robots
{
	// return chroms[c_index*pj->chromLen + 50] + 101;

	double routeDistTotal = 0.0;
	int visitedGraph[NUMBER_POINTS][NUMBER_POINTS];
	int verifyGraph[NUMBER_POINTS][NUMBER_POINTS];

	for (int x = 0; x < NUM_PERMUTATIONS; x++) {
		// Reset Visited Graph to unvisited on all edges between evaluations --
		for (int i = 0; i < NUMBER_POINTS; i++) {
			for (int j = 0; j < NUMBER_POINTS; j++) {
				visitedGraph[i][j] = 1.0;  // unvisited
				verifyGraph[i][j]  = 0;    // unvisited
			}
		}

		// get random order
		int myOrder[5] = {0, 1, 2, 3, 4};
		int temp       = 0;
		for (int y = 0; y < 5; y++) {
			temp  = myOrder[y];
			int r = y;  // int r = Rnd(0, 4); // changing this to avoid host function call in
			            // device code. Permutations/deadheading not addressed right now.
			myOrder[y] = myOrder[r];
			myOrder[r] = temp;
		}

		double robotDists[5];

		int robot1 = 0, robot2 = 0, robot3 = 0, robot4 = 0, robot5 = 0;

		// return pj->chrom[0] + 100;

		getRobot12_device(pj, &robot1, &robot2, &robot3, &robot4, &robot5, chroms, c_index);

		// int dummyvar = pj->chromLen;
		// return robot1;

		// 2 robots are connected, return a low fitness.
		if (robot2 - robot1 <= 2 || (robot2 - robot1) >= pj->chromLen - 2 || robot3 - robot2 <= 2 ||
		    robot3 - robot2 >= pj->chromLen - 2 || robot4 - robot3 <= 2 ||
		    robot4 - robot3 >= pj->chromLen - 2 || robot5 - robot4 <= 2 ||
		    robot5 - robot4 >= pj->chromLen - 2 || robot5 - robot1 >= pj->chromLen - 2 ||
		    robot1 == 1) {
			return 900000;
		}

		// return 789;

		int currentPoint1, currentPoint2, currentPoint3, currentPoint4, currentPoint5,
		    currentPoint6;
		double robot1Dis, robot2Dis, robot3Dis, robot4Dis, robot5Dis;

		// return 789;
		// printf("\n-----\n");
		for (int y = 0; y < 5; y++) {
			if (myOrder[y] == 0) {
				currentPoint1 =
				    getStartPoint(pj, (robot1 + 1) % pj->chromLen, (robot1 + 2) % pj->chromLen,
				                  visitedGraph, chroms, c_index);

				// route distance from robot1+1 to robot2
				Tuple r1;
				// r1 = routeDistance(pj, robot1 + 1, robot2, &currentPoint1);
				r1 = PhenoRouteGet(pj, robot1 + 1, robot2, &currentPoint1, visitedGraph,
				                   verifyGraph, chroms, c_index);

				robot1Dis = r1.dist;
			} else if (myOrder[y] == 1) {
				currentPoint2 =
				    getStartPoint(pj, (robot2 + 1) % pj->chromLen, (robot2 + 2) % pj->chromLen,
				                  visitedGraph, chroms, c_index);
				// route distance from robot2+1 to robot3

				Tuple r2;
				// r2 = routeDistance(pj, robot2 + 1, robot3, &currentPoint2);
				r2        = PhenoRouteGet(pj, robot2 + 1, robot3, &currentPoint2, visitedGraph,
                                   verifyGraph, chroms, c_index);
				robot2Dis = r2.dist;

			} else if (myOrder[y] == 2) {
				currentPoint3 =
				    getStartPoint(pj, (robot3 + 1) % pj->chromLen, (robot3 + 2) % pj->chromLen,
				                  visitedGraph, chroms, c_index);
				// route distance from robot3+1 to robot4

				Tuple r3;
				// r3 = routeDistance(pj, robot3 + 1, robot4, &currentPoint3);
				r3        = PhenoRouteGet(pj, robot3 + 1, robot4, &currentPoint3, visitedGraph,
                                   verifyGraph, chroms, c_index);
				robot3Dis = r3.dist;

			} else if (myOrder[y] == 3) {
				currentPoint4 =
				    getStartPoint(pj, (robot4 + 1) % pj->chromLen, (robot4 + 2) % pj->chromLen,
				                  visitedGraph, chroms, c_index);
				// route distance from robot1+1 to robot2
				Tuple r4;

				// r4 = routeDistance(pj, robot4 + 1, robot5, &currentPoint4);
				r4        = PhenoRouteGet(pj, robot4 + 1, robot5, &currentPoint4, visitedGraph,
                                   verifyGraph, chroms, c_index);
				robot4Dis = r4.dist;

			} else if (myOrder[y] == 4) {
				currentPoint5 =
				    getStartPoint(pj, (robot5 + 1) % pj->chromLen, (robot5 + 2) % pj->chromLen,
				                  visitedGraph, chroms, c_index);
				// route distance from robot5 to chromLen, part 1

				robot5Dis = 0;

				Tuple r5part1;
				int* fullRoute5part1;
				bool part1Flag = false;
				bool part2Flag = false;
				// if robot5 is the last element in the chromosome, there won't be a part1
				if (robot5 != pj->chromLen - 1) {
					// r5part1 = routeDistance(pj, (robot5 + 1) % pj->chromLen, pj->chromLen,
					// &currentPoint5);
					r5part1 =
					    PhenoRouteGet(pj, (robot5 + 1) % pj->chromLen, pj->chromLen, &currentPoint5,
					                  visitedGraph, verifyGraph, chroms, c_index);
					robot5Dis += r5part1.dist;
					fullRoute5part1 = r5part1.route;
					part1Flag       = true;
				}

				Tuple r5part2;
				int* fullRoute5part2;

				// if robot1 is the first element in the chromosome, there won't be a part2
				if (robot1 != 0) {
					// currentPoint6 = getStartPoint(pj, 0, 1);

					// route distance from 0 to robot1, part 2
					// r5part2 = routeDistance(pj, 0, robot1, &currentPoint5);
					r5part2 = PhenoRouteGet(pj, 0, robot1, &currentPoint5, visitedGraph,
					                        verifyGraph, chroms, c_index);
					robot5Dis += r5part2.dist;
					fullRoute5part2 = r5part2.route;
					part2Flag       = true;
				}

			} else {
				printf("Something went wrong with route ordering...");
			}
		}
		/*
		int sum = 0;
		for (int i = 0; i < NUMBER_POINTS; i++)
		{
		    for (int j = 0; j < NUMBER_POINTS; j++)
		    {
		        if (graph[i*NUMBER_EDGES + j] > 0)
		        {
		            sum += graph[i*NUMBER_EDGES + j];
		        }
		    }
		}

		if ((robot1Dis + robot2Dis + robot3Dis + robot4Dis + robot5Dis) < sum)
		{
		    return 900000;
		}*/

		// printf("\n-----\n");

		double longestRoute = 0;
		if (robot1Dis > longestRoute) longestRoute = robot1Dis;
		if (robot2Dis > longestRoute) longestRoute = robot2Dis;
		if (robot3Dis > longestRoute) longestRoute = robot3Dis;
		if (robot4Dis > longestRoute) longestRoute = robot4Dis;
		if (robot5Dis > longestRoute) longestRoute = robot5Dis;

		// return longestRoute;

		routeDistTotal += longestRoute;

		// routeDistTotal += robot1Dis + robot2Dis + robot3Dis + robot4Dis + robot5Dis;	//For
		// non min-max problem

		// Verify routes are a complete covering
		for (int i = 0; i < NUMBER_POINTS; i++) {
			for (int j = 0; j < NUMBER_POINTS; j++) {
				if (graph[i * NUMBER_POINTS + j] > 0 &&
				    verifyGraph[i][j] ==
				        0)  // a valid edge was not visited, return very low fitness
				{
					return 900000;
				}
			}

			// delete[] visitedGraph[i];
			// delete[] verifyGraph[i];
		}
		// delete[] visitedGraph;
		// delete[] verifyGraph;
	}

	double myFitness = (routeDistTotal / NUM_PERMUTATIONS);

	// printf("eval ending \n");
	return myFitness;
}

double TSPDist(ChromType c1, ChromType c2) {
	double xd, yd;
	xd = (double) (cities[c1].x - cities[c2].x);
	yd = (double) (cities[c1].y - cities[c2].y);
	return rint(sqrt((xd * xd) + (yd * yd)));
}

void AppInitChrom(IPTR pj) {
	Shuffle(pj->chrom, pj->chromLen);

	return;
}

void AppSkipline(FILE* fp) {
	int ch;
	while ((ch = fgetc(fp)) != '\n') {
		if (ch == EOF) {
			fprintf(stderr, "AppSkipline: Premature end of file \n");
			exit(1);
		}
	}
}

void AppInit(char* appInfile, Population* p) {
	int graph_flat[NUMBER_POINTS * NUMBER_POINTS];
	int dijkstraDistTable_flat[NUMBER_POINTS * NUMBER_POINTS];
	int dijkstraRouteTable_flat[NUMBER_POINTS * NUMBER_POINTS * MAX_ROUTE_LENGTH];

	INIT_COUNT += 1;

	FILE* fp;
	if ((fp = fopen(appInfile, "r")) == NULL) {
		fprintf(stderr, "AppInit: Cannot open %s for reading\n", appInfile);
		exit(1);
	}
	int value;
	int edge_index = 0;
	int index      = 0;
	for (int r = 0; r < NUMBER_POINTS; r++) {
		for (int c = 0; c < NUMBER_POINTS; c++) {
			if (c != NUMBER_POINTS - 1) {
				fscanf(fp, "%d,", &value);
			} else {
				fscanf(fp, "%d\n", &value);
			}
			graph_h[r][c]     = value;
			graph_flat[index] = value;
			index++;

			// printf("graph value %d inserted", graph[r][c]);

			if (value > 0 && r < c) {
				edges_h[edge_index].start = r;
				edges_h[edge_index].end   = c;
				edgeValues[edge_index]    = value;
				edge_index++;
				totalLength += value;
			}
		}
	}
	p->chromLength = edge_index + 5;
	printf("Setting the chromosome length to %d, based on number of edges in %s\n", p->chromLength,
	       appInfile);
	// initialize dijkstra table

	hipMemset(dijkstraDistTable, -1, NUMBER_POINTS * NUMBER_POINTS * sizeof(int));
	hipMemset(dijkstraRouteTable, -1,
	           NUMBER_POINTS * NUMBER_POINTS * MAX_ROUTE_LENGTH * sizeof(int));

	// copy arrays to to memory the device code can read
	hipMemcpyToSymbol(HIP_SYMBOL(edges), edges_h, NUMBER_EDGES * sizeof(Edges));
	hipMemcpyToSymbol(HIP_SYMBOL(graph), graph_flat, NUMBER_POINTS * sizeof(int) * NUMBER_POINTS);

	// for (int i = 0; i < NUMBER_POINTS; i++) {
	// 	hipMemcpyToSymbol(HIP_SYMBOL(graph[i]), graph_h[i], NUMBER_POINTS * sizeof(int));
	// 	hipMemcpyToSymbol(HIP_SYMBOL(dijkstraDistTable[i]), dijkstraDistTable_h[i],
	// 	                   NUMBER_POINTS * sizeof(int));
	// }

	printf("appinit finished\n");

	// DIJKSTRA_COUNT = 0;
}

void AppInitTSP(char* appInfile, Population* p) {
	int i, c, tmp;
	FILE* fp;
	cities = (Coords*) calloc((size_t) p->chromLength, sizeof(Coords));
	if ((fp = fopen(appInfile, "r")) == NULL) {
		fprintf(stderr, "AppInit: Cannot open %s for reading\n", appInfile);
		exit(1);
	}
	while (isalpha(c = fgetc(fp))) {  // skip header
		AppSkipline(fp);
	}
	ungetc(c, fp);

	// read coords
	for (i = 0; i < p->chromLength; i++) {
		fscanf(fp, "%d %f %f", &tmp, &(cities[i].x), &(cities[i].y));
	}
	//  for(i = 0; i < p->chromLength; i++){
	//    fprintf(stdout, "%d %f %f \n", i, (cities[i].x), (cities[i].y));
	//  }
}

// route construct
//__device__ __host__
void PhenoRoutePrint(IPTR pj, int posStart, int posEnd, FILE* fp, int* currentPoint,
                     int visitedGraph[NUMBER_POINTS][NUMBER_POINTS]) {
	// printf("PhenoRoutePrint\n");
	int dummyRoute[MAX_ROUTE_LENGTH];
	int dummyRoute2[MAX_ROUTE_LENGTH];
	for (int i = 0; i < MAX_ROUTE_LENGTH; i++) {
		dummyRoute[i]  = -1;
		dummyRoute2[i] = -1;
	}

	// Print dijkstra count
	// fprintf(fp, "\nDijkstra count: %d %d\n", DIJKSTRA_COUNT, INIT_COUNT);

	// first point
	fprintf(fp, "%d,", *currentPoint);

	// first edge
	if (*currentPoint == edges_h[pj->chrom[posStart]].start) {
		*currentPoint = edges_h[pj->chrom[posStart]].end;
		fprintf(fp, "%d,", *currentPoint);
	} else if (*currentPoint == edges_h[pj->chrom[posStart]].end) {
		*currentPoint = edges_h[pj->chrom[posStart]].start;
		fprintf(fp, "%d,", *currentPoint);
	}

	// calcluate the distance of robot 1 from 2nd edge to robot 2
	for (int i = posStart + 1; i < posEnd; i++) {
		int dissta =
		    Dijkstra_host(*currentPoint, edges_h[pj->chrom[i]].start, dummyRoute, visitedGraph);
		int disend =
		    Dijkstra_host(*currentPoint, edges_h[pj->chrom[i]].end, dummyRoute2, visitedGraph);
		if (dissta < disend) {
			if (dissta != 0) {
				for (int j = MAX_ROUTE_LENGTH - 1; j >= 0; j--) {
					if (dummyRoute[j] != -1) { fprintf(fp, "%d,", dummyRoute[j]); }
				}
				fprintf(fp, "%d,", edges_h[pj->chrom[i]].start);
			}
			*currentPoint = edges_h[pj->chrom[i]].end;
			fprintf(fp, "%d,", *currentPoint);
		} else {
			if (disend != 0) {
				// fprintf(fp, "(");
				for (int j = MAX_ROUTE_LENGTH - 1; j >= 0; j--) {
					if (dummyRoute2[j] != -1) { fprintf(fp, "%d,", dummyRoute2[j]); }
				}
				// fprintf(fp, ")");
				fprintf(fp, "%d,", edges_h[pj->chrom[i]].end);
			}
			*currentPoint = edges_h[pj->chrom[i]].start;
			fprintf(fp, "%d,", *currentPoint);
		}
		for (int j = 0; j < MAX_ROUTE_LENGTH; j++) {
			dummyRoute[j]  = -1;
			dummyRoute2[j] = -1;
		}
	}

	// printf("PhenoRoutePrint ending\n");
}

// 2 robots
//__global__
void PhenoPrint(FILE* fp, IPTR pop, Population* p)  // modified
{
	int i;

	int visitedGraph[NUMBER_POINTS][NUMBER_POINTS];
	int verifyGraph[NUMBER_POINTS][NUMBER_POINTS];

	for (int i = 0; i < NUMBER_POINTS; i++) {
		for (int j = 0; j < NUMBER_POINTS; j++) {
			visitedGraph[i][j] = 1;  // unvisited
			verifyGraph[i][j]  = 0;  // unvisited
		}
	}

	fprintf(fp, "%i ", (int) pop[p->maxi].objfunc);

	for (i = 0; i < p->chromLength; i++) {
		if (pop[p->maxi].chrom[i] >= NUMBER_EDGES) {
			fprintf(fp, " Robot-%d=>", pop[p->maxi].chrom[i]);
		} else {
			fprintf(fp, "(E%d%d)", edges_h[pop[p->maxi].chrom[i]].start,
			        edges_h[pop[p->maxi].chrom[i]].end);
		}
	}

	fprintf(fp, "\n");

	// print points only
	int robot1 = 0, robot2 = 0, robot3 = 0, robot4 = 0, robot5 = 0;
	getRobot12_host(pop, &robot1, &robot2, &robot3, &robot4, &robot5);

	// if 2 robots are connected, do not attempt to print as an error will occur.  --
	if (robot2 - robot1 <= 2 || (robot2 - robot1) >= p->chromLength - 2 || robot3 - robot2 <= 2 ||
	    robot3 - robot2 >= p->chromLength - 2 || robot4 - robot3 <= 2 ||
	    robot4 - robot3 >= p->chromLength - 2 || robot5 - robot4 <= 2 ||
	    robot5 - robot4 >= p->chromLength - 2 || robot5 - robot1 >= p->chromLength - 2) {
		fprintf(fp, "Illegal Arrangement");
		return;
	}

	int currentPoint1 = getStartPoint_host(pop, (robot1 + 1) % pop->chromLen,
	                                       (robot1 + 2) % pop->chromLen, visitedGraph);
	// route distance from robot1+1 to robot2
	fprintf(fp, "Robot-1=>");
	PhenoRoutePrint(pop, robot1 + 1, robot2, fp, &currentPoint1, visitedGraph);

	int currentPoint2 = getStartPoint_host(pop, (robot2 + 1) % pop->chromLen,
	                                       (robot2 + 2) % pop->chromLen, visitedGraph);
	// route distance from robot2+1 to robot3
	fprintf(fp, "Robot-2=>");
	PhenoRoutePrint(pop, robot2 + 1, robot3, fp, &currentPoint2, visitedGraph);

	int currentPoint3 = getStartPoint_host(pop, (robot3 + 1) % pop->chromLen,
	                                       (robot3 + 2) % pop->chromLen, visitedGraph);
	// route distance from robot3+1 to robot4
	fprintf(fp, "Robot-3=>");
	PhenoRoutePrint(pop, robot3 + 1, robot4, fp, &currentPoint3, visitedGraph);

	int currentPoint4 = getStartPoint_host(pop, (robot4 + 1) % pop->chromLen,
	                                       (robot4 + 2) % pop->chromLen, visitedGraph);
	// route distance from robot4+1 to robot5
	fprintf(fp, "Robot-4=>");
	PhenoRoutePrint(pop, robot4 + 1, robot5, fp, &currentPoint4, visitedGraph);

	int currentPoint5 = getStartPoint_host(pop, (robot5 + 1) % pop->chromLen,
	                                       (robot5 + 2) % pop->chromLen, visitedGraph);
	// route distance from robot5 to chromLen, part 1
	fprintf(fp, "\nRobot-5=>");
	// in case robot5 is the last element in the chromosome
	if (robot5 != pop->chromLen - 1) {
		PhenoRoutePrint(pop, (robot5 + 1) % pop->chromLen, pop->chromLen, fp, &currentPoint5,
		                visitedGraph);
	}
	// route distance from 0 to robot1, part 2
	PhenoRoutePrint(pop, 0, robot1, fp, &currentPoint5, visitedGraph);

	fprintf(fp, "\n");

	/*
	//Verify routes are a complete covering
	for (int i = 0; i < NUMBER_POINTS; i++)
	{
	    for (int j = 0; j < NUMBER_POINTS; j++)
	    {
	        if (graph_h[i][j] > 0 && verifyGraph[i][j] == 0)	//a valid edge was not visited,
	return very low fitness
	        {
	            fprintf(fp," ^ ILLEGAL ROUTE - An Edge was Skipped\n");
	        }
	    }

	    //delete[] visitedGraph[i];
	    //delete[] verifyGraph[i];
	}*/

	/*
	for (int i = 0; i < NUMBER_POINTS; i++)
	{
	    delete[] visitedGraph[i];
	    delete[] verifyGraph[i];
	}

	delete[] visitedGraph;
	delete[] verifyGraph;*/
}

// function to return route and cost of one section of genome (1 robot's path)
__device__ Tuple PhenoRouteGet(IPTR pj, int posStart, int posEnd, int* currentPoint,
                               int visitedGraph[NUMBER_POINTS][NUMBER_POINTS],
                               int verifyGraph[NUMBER_POINTS][NUMBER_POINTS], int* chroms,
                               int c_index) {
	// printf("phenoRouteGet\n");
	Tuple myTuple;

	// printf("posStart: %d\n", posStart);
	// printf("posEnd: %d\n", posEnd);
	// printf("curr: %d\n", currentPoint);

	if (posStart == posEnd) {
		myTuple.dist = 900000000;
		return myTuple;
	}

	int myRoute[MAX_ROUTE_LENGTH];
	int routeIndex = 0;

	int dummyRoute[MAX_ROUTE_LENGTH];
	int dummyRoute2[MAX_ROUTE_LENGTH];
	for (int i = 0; i < MAX_ROUTE_LENGTH; i++) {
		dummyRoute[i]  = -1;
		dummyRoute2[i] = -1;
		myRoute[i]     = -1;
		// myTuple.route[i] = -1;
	}

	// first point
	// fprintf(fp, "%d,", *currentPoint);
	myRoute[routeIndex] = *currentPoint;
	routeIndex++;

	// first edge
	if (*currentPoint == edges[chroms[pj->chromLen * c_index + posStart]].start) {
		*currentPoint = edges[chroms[pj->chromLen * c_index + posStart]].end;
		// fprintf(fp, "%d,", *currentPoint);
		myRoute[routeIndex] = *currentPoint;
		routeIndex++;
	} else if (*currentPoint == edges[chroms[pj->chromLen * c_index + posStart]].end) {
		*currentPoint = edges[chroms[pj->chromLen * c_index + posStart]].start;
		// fprintf(fp, "%d,", *currentPoint);
		myRoute[routeIndex] = *currentPoint;
		routeIndex++;
	}

	// calcluate the distance of robot 1 from 2nd edge to robot 2

	// printf("phenoRouteGet just before posStart looop\n");
	for (int i = posStart + 1; i < posEnd; i++) {
		int dissta = Dijkstra(*currentPoint, edges[chroms[pj->chromLen * c_index + i]].start,
		                      dummyRoute, visitedGraph);
		int disend = Dijkstra(*currentPoint, edges[chroms[pj->chromLen * c_index + i]].end,
		                      dummyRoute2, visitedGraph);
		if (dissta < disend) {
			///////
			// myTuple.dist = dissta;
			// return myTuple;
			///////
			if (dissta != 0) {
				for (int j = MAX_ROUTE_LENGTH - 1; j >= 0; j--) {
					if (dummyRoute[j] != -1) {
						// fprintf(fp, "%d,", dummyRoute[j]);
						myRoute[routeIndex] = dummyRoute[j];
						routeIndex++;
					}
				}
				// fprintf(fp, "%d,", edges[pj->chrom[i]].start);
				myRoute[routeIndex] = edges[chroms[pj->chromLen * c_index + i]].start;
				routeIndex++;
			}
			*currentPoint = edges[chroms[pj->chromLen * c_index + i]].end;
			// fprintf(fp, "%d,", *currentPoint);
			myRoute[routeIndex] = *currentPoint;
			routeIndex++;
		} else {
			///////
			// myTuple.dist = disend;
			// return myTuple;
			///////

			if (disend != 0) {
				// fprintf(fp, "(");
				for (int j = MAX_ROUTE_LENGTH - 1; j >= 0; j--) {
					if (dummyRoute2[j] != -1) {
						// fprintf(fp, "%d,", dummyRoute2[j]);
						myRoute[routeIndex] = dummyRoute2[j];
						routeIndex++;
					}
				}
				// fprintf(fp, ")");
				// fprintf(fp, "%d,", edges[pj->chrom[i]].end);
				myRoute[routeIndex] = edges[chroms[pj->chromLen * c_index + i]].end;
				routeIndex++;
			}
			*currentPoint = edges[chroms[pj->chromLen * c_index + i]].start;
			// fprintf(fp, "%d,", *currentPoint);
			myRoute[routeIndex] = *currentPoint;
			routeIndex++;
		}
		for (int j = 0; j < MAX_ROUTE_LENGTH; j++) {
			dummyRoute[j]  = -1;
			dummyRoute2[j] = -1;
		}
	}

	// printf("phenoRouteGet copying to myTuple route\n");
	for (int i = 0; i < MAX_ROUTE_LENGTH; i++) {
		// printf("myRoute[i]: %d\n", myRoute[i]);
		myTuple.route[i] = myRoute[i];
	}

	// printf("phenoRouteGet about to clean duplicates\n");

	// clean route of duplicates
	for (int x = 0; x < 2; x++) {
		for (int i = 1; i < MAX_ROUTE_LENGTH; i++) {
			if (myTuple.route[i - 1] == myTuple.route[i]) {
				for (int j = i; j < MAX_ROUTE_LENGTH - 1; j++) {
					myTuple.route[j] = myTuple.route[j + 1];
				}
			}
		}
	}

	/*
	printf("\n\n");
	for (int i = 0; i < MAX_ROUTE_LENGTH; i++)
	{
	    if (myTuple.route[i] != -1)
	    {
	        printf("%d,", myTuple.route[i]);
	    }
	}
	printf("\n\n");*/

	double myDist = 0.0;

	// printf("phenoRouteGet about to calculate cost\n");

	// calculate cost of route
	for (int i = 0; i < MAX_ROUTE_LENGTH - 1; i++) {
		if (myTuple.route[i] != -1 && myTuple.route[i + 1] != -1) {
			myDist += graph[myTuple.route[i] * NUMBER_POINTS + myTuple.route[i + 1]] *
			          visitedGraph[myTuple.route[i]][myTuple.route[i + 1]];  // Add distance
			visitedGraph[myTuple.route[i]][myTuple.route[i + 1]] =
			    DEADHEADING_RATIO;  // Mark edge as visited
			// printf("\n | %d, %d | \n", myTuple.route[i], myTuple.route[i + 1]);
			visitedGraph[myTuple.route[i + 1]][myTuple.route[i]] = DEADHEADING_RATIO;
			verifyGraph[myTuple.route[i]][myTuple.route[i + 1]]  = 1;
			verifyGraph[myTuple.route[i + 1]][myTuple.route[i]]  = 1;
		}
	}

	myTuple.dist = myDist;

	// printf("phenoRouteGet ending\n");
	return myTuple;
}

// Single depot
/*
void PhenoPrint1R(FILE *fp, IPTR pop, Population *p)
{
    int i;

    fprintf(fp, "%i ", (int)pop[p->maxi].objfunc);

    for (i = 0; i < p->chromLength; i++) {
        if (pop[p->maxi].chrom[i] >= NUMBER_EDGES) {
            fprintf(fp, " Robot-%d=>", pop[p->maxi].chrom[i]);
        }
        else {
            fprintf(fp, "(E%d%d)", edges[pop[p->maxi].chrom[i]].start,
edges[pop[p->maxi].chrom[i]].end);
        }
    }

    fprintf(fp, "\n");

    //print points only
    int currentPoint1 = getStartPoint(pop, 0, 1);
    //route distance from robot1+1 to robot2
    PhenoRoutePrint(pop, 0, pop->chromLen, fp, &currentPoint1);

    fprintf(fp, "\n");
}*/

void TourPrint(FILE* fp, IPTR pj, char* name) {
	int i;
	fprintf(fp, "%s :", name);
	for (i = 0; i < pj->chromLen; i++) { fprintf(fp, "%i ", pj->chrom[i]); }
	fprintf(fp, "\n");
}

void InitPhenoPrint(IPTR pj, char* fname, Population* p) {
	FILE* fp;
	int i;

	if ((fp = fopen(fname, "w")) == NULL) {
		fprintf(stderr, "InitPhenoPrint: Cannot open %s for writing\n", fname);
		exit(1);
	}
	fprintf(fp, "%i \nTourLength ", (p->chromLength + 1) * 2 + 1);
	for (i = 0; i < p->chromLength; i++) { fprintf(fp, "x y "); }
	fprintf(fp, "x y ");
	fprintf(fp, "\n");
	fclose(fp);
}

void RemoveCrossings(IPTR pj, int start, int end) {
	int i, j;
	int lchrom;
	IPTR tmp;
	ChromType city1Next, city2Prev;

	lchrom = pj->chromLen;
	tmp    = AllocateIndividuals(1, lchrom);

	IndividualCopy(pj, tmp);
	for (i = start; i < end; i++) {
		for (j = (i + 3) % lchrom; j != i; j = ((j + 1) % lchrom)) {
			city1Next = (i + 1) % lchrom;
			city2Prev = (j + lchrom - 1) % lchrom;
			if ((TSPDist(tmp->chrom[i], tmp->chrom[city1Next]) +
			     TSPDist(tmp->chrom[j], tmp->chrom[city2Prev])) >
			    (TSPDist(tmp->chrom[i], tmp->chrom[city2Prev]) +
			     TSPDist(tmp->chrom[j], tmp->chrom[city1Next]))) {
				Reverse(tmp, lchrom, city1Next, city2Prev);
			}
		}
	}
	IndividualCopy(tmp, pj);
	free(tmp->chrom);
	free(tmp->backup);
	free(tmp);
	return;
}

void Reverse(IPTR tmp, int lchrom, ChromType c1, ChromType c2) {
	SwapChromType(&(tmp->chrom[c1]), &(tmp->chrom[c2]));
	/****
	do {
	  SwapChromType(&(tmp->chrom[c1]), &(tmp->chrom[c2]));
	  c2 = (c2 + lchrom - 1) % lchrom;
	  c1 = (c1 + 1) % lchrom;
	} while (c2 != c1 && ((c1 + lchrom - 1 ) % lchrom) != c2);
	****/
	return;
}
